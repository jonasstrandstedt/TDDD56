#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>

const int N = 16; 
const int blocksize = 16; 

__global__ 
void simple(float *c) 
{
	//c[threadIdx.x] = threadIdx.x;
	c[threadIdx.x] = sqrt(c[threadIdx.x]);
}

int main()
{
	float *c = new float[N];

	for(int i = 0; i < N; ++i) {
		c[i] = i*2;
	}

	float *cd;
	const int size = N*sizeof(float);
	
	hipMalloc( (void**)&cd, size );
	hipMemcpy( cd, c, size, hipMemcpyHostToDevice ); 


	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	simple<<<dimGrid, dimBlock>>>(cd);
	hipDeviceSynchronize();
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
	hipFree( cd );
	
	for (int i = 0; i < N; i++)
		printf("%.8f ", c[i]);
	printf("\n");
	for(int i = 0; i < N; ++i) {
		printf("%.8f ",sqrt(i*2));
	}
	printf("\n");

	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
