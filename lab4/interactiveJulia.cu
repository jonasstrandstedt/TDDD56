#include "hip/hip_runtime.h"
// Ingemars rewrite of the julia demo, integrating the OpenGL parts.
// The CUDA parts are - intentionally - NOT rewritten, and have some
// serious performance problems. Find the problems and make this a�
// decently performing CUDA program.

// Compile with
// nvcc -lglut -lGL interactiveJulia.cu -o interactiveJulia

#include <GL/glut.h>
#include <GL/gl.h>
#include <stdio.h>
#include "milli.h"


hipEvent_t myEventStart;
hipEvent_t myEventStop;
unsigned char *dev_bitmap;

// Image data
	unsigned char	*pixels;
	int	 gImageWidth, gImageHeight;

// Init image data
void initBitmap(int width, int height)
{
	pixels = (unsigned char *)malloc(width * height * 4);
	gImageWidth = width;
	gImageHeight = height;
}

#define DIM 1024

// Complex number class
struct hipComplex
{
    float   r;
    float   i;
    
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    
    __device__ float magnitude2( void )
    {
        return r * r + i * i;
    }
    
    __device__ hipComplex operator*(const hipComplex& a)
    {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    
    __device__ hipComplex operator+(const hipComplex& a)
    {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ int julia( int x, int y, float r, float im)
{
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

//    hipComplex c(-0.8, 0.156);
    hipComplex c(r, im);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++)
    {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return i;
    }

    return i;
}

__global__ void kernel( unsigned char *ptr, float r, float im)
{
    // map from blockIdx to pixel position
    //int x = blockIdx.x;
    //int y = blockIdx.y;
    //int offset = x + y * gridDim.x;

    int x = blockIdx.x * blockDim.x + threadIdx.x;    
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int grid_width = gridDim.x * blockDim.x;

    //get the global index 
    int offset = y * grid_width + x;

    // now calculate the value at that position
    int juliaValue = julia( x, y, r, im );
    ptr[offset*4 + 0] = 255 * juliaValue/200;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

float theReal, theImag;

// Compute CUDA kernel and display image
void Draw()
{


    hipEventRecord(myEventStart, 0);
    hipEventSynchronize(myEventStart);
	
    const int blocksize = 16; 
    const int grid_N = DIM / blocksize;

    dim3 dimBlock( blocksize, blocksize );
    dim3 dimGrid( grid_N, grid_N );

	kernel<<<dimGrid, dimBlock>>>( dev_bitmap, theReal, theImag);
	hipDeviceSynchronize();
	hipMemcpy( pixels, dev_bitmap, gImageWidth*gImageHeight*4, hipMemcpyDeviceToHost );


    hipEventRecord(myEventStop, 0);
    hipEventSynchronize(myEventStop);


    float theTime;
    hipEventElapsedTime(&theTime, myEventStart, myEventStop);
    printf("Cuda time: %f\n", theTime);
	

    // Dump the whole picture onto the screen.	
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );
	glDrawPixels( gImageWidth, gImageHeight, GL_RGBA, GL_UNSIGNED_BYTE, pixels );
	glutSwapBuffers();
}

void MouseMovedProc(int x, int y)
{
	theReal = -0.5 + (float)(x-400) / 500.0;
	theImag = -0.5 + (float)(y-400) / 500.0;
	//printf("real = %f, imag = %f\n", theReal, theImag);
	glutPostRedisplay ();
}

// Main program, inits
int main( int argc, char** argv) 
{

    hipEventCreate(&myEventStart);
    hipEventCreate(&myEventStop);

	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA );
	glutInitWindowSize( DIM, DIM );
	glutCreateWindow("CUDA on live GL");
	glutDisplayFunc(Draw);
	glutPassiveMotionFunc(MouseMovedProc);
	
	initBitmap(DIM, DIM);

    hipMalloc( &dev_bitmap, gImageWidth*gImageHeight*4 );
	glutMainLoop();
    hipFree( dev_bitmap );

    hipEventDestroy(myEventStart);
    hipEventDestroy(myEventStop);
}
